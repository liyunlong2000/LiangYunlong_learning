
#include <hip/hip_runtime.h>
#include<iostream>
#include <stdlib.h> 
#include <time.h>
#include <sys/time.h>
using namespace std;

const int m=3024;
__global__ void mul(float* ad,float*bd,float*cd){
    int i=threadIdx.x+(blockDim.x*blockIdx.x);
    int j=threadIdx.y+(blockDim.y*blockIdx.y);
    if(i<m && j<m){
    float sum=0;
    for(int k=0;k<m;k++){
        sum+=ad[i*m+k]*bd[k*m+j];
    }
    cd[i*m+j]=sum;
    }
}
int main(){
    float *a,*b,*c,*ad,*bd,*cd;
    int total_size=m*m*sizeof(float);
    a = (float*)malloc(total_size);
    b = (float*)malloc(total_size);
    c = (float*)malloc(total_size);
    hipMalloc((void**)&ad,total_size);
    hipMalloc((void**)&bd,total_size);
    hipMalloc((void**)&cd,total_size);
    srand((unsigned)time(NULL));
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < m; j++)
        {
            a[i*m+j]=20.3;
            b[i*m+j]=5.7;
        }
        
    }
    struct timeval start, end;
    gettimeofday( &start, NULL );
    hipMemcpy(ad,a,total_size,hipMemcpyHostToDevice);
    hipMemcpy(bd,b,total_size,hipMemcpyHostToDevice);
    hipMemcpy(cd,c,total_size,hipMemcpyHostToDevice);
    dim3 blockSize(32,32);
    dim3 gridSize(m/32+1,m/32+1);
    mul<<<gridSize,blockSize>>>(ad,bd,cd);
    hipMemcpy(c,cd,total_size,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    gettimeofday( &end, NULL );

    for(int i=m-10;i<m;i++){
        for(int j=m-10;j<m;j++){
            cout<<c[i*m+j]<<" ";
        }
        cout<<endl;
    }
        cout<<"-------"<<endl;
        for(int i=0;i<10;i++){
        for(int j=0;j<10;j++){
            cout<<c[i*m+j]<<" ";
        }
        cout<<endl;
    }
    free(a);
    free(b);
    free(c);
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    cout << "total time is " << timeuse/1000 << "ms" <<endl;
    return 0;
}